#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <stdio.h>
#include <iostream>
#include <thread>
#include "cuda_probability.cuh"
#include "utils.hpp"  // For printProgress

#define CHECK_CUDA_ERROR(val) check_cuda((val), #val, __FILE__, __LINE__)
template <typename T> void check_cuda(T err, const char* const func, const char* const file, const int line) {
  if (err != hipSuccess) {
    fprintf(stderr, "CUDA error at %s:%d code=%d(%s) \"%s\" \n", file, line, static_cast<unsigned int>(err),
            hipGetErrorString(err), func);
    hipDeviceReset();
    exit(EXIT_FAILURE);
  }
}

// Add these helper functions at the start of the file
void checkGPUMemory() {
  size_t free, total;
  hipMemGetInfo(&free, &total);
  fprintf(stderr, "GPU Memory - Free: %zu MB, Total: %zu MB\n", free / (1024 * 1024), total / (1024 * 1024));
}

// Constants for CUDA execution
const int BLOCK_SIZE = 256;
const int NUM_BLOCKS = 256;

__device__ bool hasFlush(const unsigned char* hand, bool exclusive = true) {
  int suit = hand[0] & 3;
  for (int i = 1; i < 5; i++) {
    if ((hand[i] & 3) != suit) return false;
  }
  return true;  // Remove exclusivity check for now to break circular dependency
}

__device__ bool hasStraight(const unsigned char* hand, bool exclusive = true) {
  int ranks[13] = {0};
  int min_rank = 13, max_rank = 0;

  for (int i = 0; i < 5; i++) {
    int rank = (hand[i] >> 2);
    ranks[rank] = 1;
    min_rank = min(min_rank, rank);
    max_rank = max(max_rank, rank);
  }

  bool isStraight = false;

  // Check for Ace-low straight
  if (ranks[12] && ranks[0] && ranks[1] && ranks[2] && ranks[3]) {
    isStraight = true;
  }
  // Check normal straight
  else if (max_rank - min_rank == 4) {
    isStraight = true;
    for (int i = min_rank; i <= max_rank; i++) {
      if (!ranks[i]) {
        isStraight = false;
        break;
      }
    }
  }

  return isStraight && (!exclusive || !hasFlush(hand, false));
}

__device__ bool hasOnePair(const unsigned char* hand) {
  int ranks[13] = {0};
  int pairs = 0;
  for (int i = 0; i < 5; i++) {
    int rank = (hand[i] >> 2);  // Fix rank calculation
    if (++ranks[rank] == 2) pairs++;
  }
  return pairs == 1;
}

__device__ bool hasTwoPair(const unsigned char* hand) {
  int ranks[13] = {0};
  int pairs = 0;
  for (int i = 0; i < 5; i++) {
    int rank = (hand[i] >> 2);  // Fix rank calculation
    if (++ranks[rank] == 2) pairs++;
  }
  return pairs == 2;
}

__device__ bool hasThreeOfAKind(const unsigned char* hand) {
  int ranks[13] = {0};
  for (int i = 0; i < 5; i++) {
    int rank = (hand[i] >> 2);          // Fix rank calculation (same as other methods)
    ranks[rank]++;                      // Count ranks without range check
    if (ranks[rank] >= 3) return true;  // Return as soon as we find three
  }
  return false;
}

__device__ bool hasFullHouse(const unsigned char* hand) {
  int ranks[13] = {0};

  // Count all ranks first
  for (int i = 0; i < 5; i++) {
    int rank = (hand[i] >> 2);
    ranks[rank]++;
  }

  bool hasThree = false, hasTwo = false;
  for (int count : ranks) {
    if (count == 3) hasThree = true;
    if (count == 2) hasTwo = true;
  }

  return hasThree && hasTwo;
}

__device__ bool hasFourOfAKind(const unsigned char* hand) {
  int ranks[13] = {0};
  for (int i = 0; i < 5; i++) {
    int rank = (hand[i] >> 2);  // Fix rank calculation
    if (++ranks[rank] == 4) return true;
  }
  return false;
}

__device__ bool hasRoyalFlush(const unsigned char* hand) {
  if (!hasFlush(hand)) return false;
  int ranks[13] = {0};
  for (int i = 0; i < 5; i++) {
    int rank = (hand[i] >> 2);  // Fix rank calculation
    ranks[rank] = 1;
  }
  return ranks[8] && ranks[9] && ranks[10] && ranks[11] && ranks[12];
}

__device__ bool hasStraightFlush(const unsigned char* hand) {
  return hasFlush(hand, false) && hasStraight(hand, false);
}

__device__ HandType getHandType(const unsigned char* hand) {
  if (hasRoyalFlush(hand)) return HandType::RoyalFlush;
  if (hasStraightFlush(hand)) return HandType::StraightFlush;
  if (hasFourOfAKind(hand)) return HandType::FourOfAKind;
  if (hasFullHouse(hand)) return HandType::FullHouse;
  if (hasFlush(hand)) return HandType::Flush;
  if (hasStraight(hand)) return HandType::Straight;
  if (hasThreeOfAKind(hand)) return HandType::ThreeOfAKind;
  if (hasTwoPair(hand)) return HandType::TwoPair;
  if (hasOnePair(hand)) return HandType::OnePair;
  return HandType::HighCard;
}

__device__ void dealRandomHand(hiprandState* state, unsigned char* hand) {
  unsigned char deck[52];

  // Initialize deck with cards: rank << 2 | suit
  for (int i = 0; i < 52; i++) {
    unsigned char rank = (i / 4);  // Ranks 2-14
    unsigned char suit = i % 4;    // Suits 0-3
    deck[i] = (rank << 2) | suit;  // Combine into single byte
  }

  // Fisher-Yates shuffle for first 5 cards
  for (int i = 0; i < 5; i++) {
    int remaining = 52 - i;
    int j = hiprand(state) % remaining;
    hand[i] = deck[j];
    deck[j] = deck[remaining - 1];
  }
}

__device__ void dealRandomHand(unsigned char* deck, int& deckPosition, hiprandState* state, unsigned char* hand) {
  // If we need to reshuffle
  if (deckPosition >= 50) {  // 52 - 2 to ensure we always have 5 cards available
    // Reset deck
    for (int i = 0; i < 52; i++) {
      unsigned char rank = (i / 4);  // Ranks 0-12
      unsigned char suit = i % 4;    // Suits 0-3
      deck[i] = (rank << 2) | suit;  // Combine into single byte
    }
    // Fisher-Yates shuffle
    for (int i = 51; i > 0; i--) {
      int j = hiprand(state) % (i + 1);
      unsigned char temp = deck[i];
      deck[i] = deck[j];
      deck[j] = temp;
    }
    deckPosition = 0;
  }

  // Deal 5 cards from current deck position
  for (int i = 0; i < 5; i++) {
    hand[i] = deck[deckPosition++];
  }
}

__global__ void simulateHandsKernel(HandType targetType, hiprandState* states, unsigned long long* counts,
                                    int handsPerThread) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState localState = states[tid];
  unsigned char hand[5];
  unsigned long long localCount = 0;

  for (int i = 0; i < handsPerThread; i++) {
    dealRandomHand(&localState, hand);
    if (getHandType(hand) == targetType) {
      localCount++;
    }
  }

  counts[tid] = localCount;
  states[tid] = localState;
}

__global__ void initRNG(hiprandState* states, unsigned long seed) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprand_init(seed + tid, 0, 0, &states[tid]);
}

double calculateHandTypeProbabilityCUDA(HandType type, int totalHands) {
  HandTypeCounts results = calculateAllProbabilitiesCUDA(totalHands);
  return results.getProbability(type);
}

__global__ void simulateHandsKernelAllTypes(hiprandState* states, unsigned long long* counts, int handsPerThread) {
  int tid = blockIdx.x * blockDim.x + threadIdx.x;
  hiprandState localState = states[tid];
  unsigned char hand[5];
  unsigned char deck[52];
  int deckPosition = 52;  // Force initial shuffle
  unsigned long long localCounts[10] = {0};

  for (int i = 0; i < handsPerThread; i++) {
    dealRandomHand(deck, deckPosition, &localState, hand);
    HandType type = getHandType(hand);
    localCounts[static_cast<int>(type)]++;
  }

  // Write results to global memory
  for (int i = 0; i < 10; i++) {
    atomicAdd(&counts[i], localCounts[i]);
  }

  states[tid] = localState;
}

// ...existing code through device functions...

HandTypeCounts calculateAllProbabilitiesCUDA(int totalHands) {
  // Single batch, using maximum thread capacity
  const int MAX_THREADS = 65536;  // 256 blocks * 256 threads
  const int numThreads = std::min(MAX_THREADS, BLOCK_SIZE * NUM_BLOCKS);
  const int actualBlocks = (numThreads + BLOCK_SIZE - 1) / BLOCK_SIZE;
  const int handsPerThread = (totalHands + numThreads - 1) / numThreads;

  // Allocate resources
  hiprandState* d_states = nullptr;
  unsigned long long* d_counts = nullptr;
  unsigned long long* h_counts = new unsigned long long[10]();
  HandTypeCounts result;

  try {
    // Allocate GPU memory
    if (hipMalloc(&d_states, numThreads * sizeof(hiprandState)) != hipSuccess ||
        hipMalloc(&d_counts, 10 * sizeof(unsigned long long)) != hipSuccess) {
      throw std::runtime_error("Failed to allocate GPU memory");
    }

    // Initialize memory and RNG
    if (hipMemset(d_counts, 0, 10 * sizeof(unsigned long long)) != hipSuccess) {
      throw std::runtime_error("Failed to initialize GPU memory");
    }

    initRNG<<<actualBlocks, BLOCK_SIZE>>>(d_states, time(nullptr));
    if (hipGetLastError() != hipSuccess) {
      throw std::runtime_error("Failed to initialize RNG");
    }

    // Run simulation
    simulateHandsKernelAllTypes<<<actualBlocks, BLOCK_SIZE>>>(d_states, d_counts, handsPerThread);
    if (hipGetLastError() != hipSuccess) {
      throw std::runtime_error("Kernel execution failed");
    }

    // Get results
    if (hipMemcpy(h_counts, d_counts, 10 * sizeof(unsigned long long), hipMemcpyDeviceToHost) != hipSuccess) {
      throw std::runtime_error("Failed to copy results from GPU");
    }

    // Scale results if needed
    unsigned long long total = 0;
    for (int i = 0; i < 10; i++) {
      total += h_counts[i];
    }

    if (total > static_cast<unsigned long long>(totalHands)) {
      double scale = static_cast<double>(totalHands) / total;
      for (int i = 0; i < 10; i++) {
        result.counts[i] = static_cast<unsigned long long>(h_counts[i] * scale);
      }
    } else {
      for (int i = 0; i < 10; i++) {
        result.counts[i] = h_counts[i];
      }
    }
  } catch (const std::exception& e) {
    // Cleanup on error
    if (d_states) hipFree(d_states);
    if (d_counts) hipFree(d_counts);
    delete[] h_counts;
    throw;
  }

  // Normal cleanup
  hipFree(d_states);
  hipFree(d_counts);
  delete[] h_counts;

  return result;
}
